
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "hiprand/hiprand.h"
#include "hiprand/hiprand_kernel.h"

#define BLOCKS 10
#define THREADS 256

__global__ void rng_init(hiprandState_t *rng_state) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long seed = clock64() + index;
    hiprand_init(seed, index, 0, &rng_state[index]);
}

__global__ void rng_generate(double *vector, hiprandState *rng_state) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState* local_rng = &rng_state[index];
    double x = hiprand_uniform_double(local_rng);
    double y = hiprand_uniform_double(local_rng);
    if (x * x + y * y < 1) {
        vector[index] = 1; 
    } else {
        vector[index] = 0; 
    }
}

int main() {
    hiprandState_t *dev_rng_state;
    int numBytes = BLOCKS * THREADS * sizeof(double);
    double *dev_v;
    double *v = new double[BLOCKS * THREADS]; 
    for (int i = 0; i < BLOCKS * THREADS; i++) {
        v[i] = 0;
    }

    hipMalloc((void **)&dev_rng_state, BLOCKS * THREADS * sizeof(hiprandState_t));
    hipMalloc((void **)&dev_v, numBytes);

    hipMemcpy(dev_v, v, numBytes, hipMemcpyHostToDevice);
    
    rng_init<<<BLOCKS, THREADS>>>(dev_rng_state);
    rng_generate<<<BLOCKS, THREADS>>>(dev_v, dev_rng_state);
    
    hipMemcpy(v, dev_v, numBytes, hipMemcpyDeviceToHost);

    int count = 0;
    for (int i = 0; i < BLOCKS * THREADS; i++) {
        if (v[i] == 1) {
            count++;
        }
    }

    double pi = 4.0 * count / (BLOCKS * THREADS);
    printf("Estimated value of Pi: %f\n", pi);

    delete[] v;
    hipFree(dev_v);
    hipFree(dev_rng_state);
    
    hipDeviceSynchronize();

    return 0;
}
